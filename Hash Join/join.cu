
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<string>
#include<fstream>
#include<iostream>
#include <hiprand/hiprand_kernel.h>
#include <vector>
#include <sstream>



#define p 334214459
#define TABLESIZE 100000
#define maxiterations 10
#define KEYEMPTY -1
#define NOTFOUND -100

__device__
unsigned long long  table[TABLESIZE];



__device__
unsigned long long make_entry(unsigned long key, unsigned long value){
  //printf("key : %d, value : %d",key , value);
  unsigned long long ans = (key<<32)+value;
  //printf ("ans : %d ", (int)ans>>32);
  printf("\n");
  return ans;
}

__device__ unsigned getkey(unsigned long long entry){
return (entry)>>32;
}

__device__ unsigned getvalue(unsigned long long entry){
  return (entry & 0xffffffff) ;
}

__device__
unsigned hash_function_1(unsigned key){
   int a1 = 5;
   int b1 = 2;
   return (((a1*key+b1)%p)%TABLESIZE);
}

__device__
unsigned hash_function_2(unsigned key){
   int a1 = 13;
   int b1 = 7;
   return (((a1*key+b1)%p)%TABLESIZE);
}



__global__
void join(int *Table_B,int *Table_C,int width_c,int width,int height){
  int index = blockIdx.x * blockDim.x +threadIdx.x;
  unsigned long primkey = Table_B[index*width+0];
   //printf("primkey : %lu \n",primkey);
  unsigned long value = Table_B[index*width+1];
  unsigned location_1 = hash_function_1(primkey);
  unsigned location_2 = hash_function_2(primkey);
  unsigned long long entry;
  if (getkey(entry = table[location_1])!= primkey)
    if (getkey(entry = table[location_2])!= primkey){
        entry = make_entry(0,NOTFOUND);
    }
 // printf("entry of primkey %lu:%llu \n",primkey,entry);
  //printf("key from hash table of primkey %lu: %d\n",primkey,getkey(entry));
  Table_C[index*width_c+0]=getkey(entry);
  Table_C[index*width_c+1]=getvalue(entry);
  //printf("key from hash table of primkey %lu: %d\n",primkey,getvalue(entry));
  Table_C[index*width_c+2] = value;
  //printf("value from hash table of primkey %lu: %d\n",primkey,value); 
  for(int l =0 ;l<3 ;l++){
    //printf("index : %d,Table: %d  ",index,Table_C[index*width_c+l]);
  }
  }

__global__
void hash(int *Table_A, int width, int height){
  
  int index = blockIdx.x * blockDim.x +threadIdx.x;
    unsigned long key = Table_A[index*width+0];
    unsigned long value = Table_A[index*width+1]; 
    unsigned long long entry = make_entry(key,value);
    //printf("entry: %d",entry);
    unsigned location = hash_function_1(key);
    unsigned k = key;
    for (int its = 0; its<maxiterations; its++){
    entry = atomicExch(&table[location], entry);
    key = getkey(entry);
    if (key == 0) {
      //printf("key: %lu table: %llu \n",k,table[location]);
      return;}
    unsigned location1 = hash_function_1(key);
    unsigned location2 = hash_function_2(key);
    if (location == location1)
     location = location2;
    else if (location == location2)
     location = location1;
    };
    printf("chain was too long");
    return ;
}



int main()
{

    int *Table_A;
    int *Table_B;
    int *Table_C;

    int width = 2;
    int width_c = 3;
    int height_a =  2500 ;//2500;
    int height_b =  1500000 ;//1500000;
    int num1=1;
    int num2 =101;
    int num3 = 201;
    int count =0;

     hipMallocManaged(&Table_A, width * height_a * sizeof(unsigned long));
     hipMallocManaged(&Table_B, width * height_b * sizeof(unsigned long));
     hipMallocManaged(&Table_C, width_c * height_b * sizeof(unsigned long));
      std::fstream fin;
      fin.open("table_a.csv", std::ios::in);
      std::string line, word;
      int i=0;
      char delimiter;
      int temp[20];
      delimiter = ',';
      //std::string l = "hi how are you";
      while (getline(fin, line,'\n')){
        std::stringstream s(line);
        while (getline(s, word,','))
        {
          Table_A[i]=stoi(word);
          //std::cout<<"table_a: "<<Table_A[i]<<"\n";
          i++;
      }
    }
    fin.close();
    int k=0;
    fin.open("table_b.csv", std::ios::in);
    while (getline(fin, line,'\n')){
      std::stringstream s(line);
      while (getline(s, word,','))
      {
        Table_B[k]=stoi(word);
        //std::cout<<"table_b: "<<Table_B[k]<<"\n";
        k++;
      }
  }
  fin.close();


     hash<<<3,1024>>>(Table_A, width, height_a);
     hipDeviceSynchronize();
     join<<<1465,1024>>>(Table_B,Table_C,width_c,width,height_b);
     hipDeviceSynchronize();
    for(int i = 0;i<height_b;i++){
      for(int l =0 ;l<width_c ;l++){
      printf(" %d ",Table_C[i*width_c+l]);
    } 
    printf("\n");
  }
    printf("exit ");
}
